#include <concurrencysal.h>
#include <memory>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/marching_cubes.h>

#include <tiny-cuda-nn/common.h>

#include <synerfgine/display.cuh>
#include <synerfgine/common.cuh>
#include <imgui/imgui.h>

namespace sng {

bool Display::m_is_init = false;

void glfw_error_callback(int error, const char* description) {
	tlog::error() << "GLFW error #" << error << ": " << description;
}

GLFWwindow* Display::init_window(int resw, int resh, const std::string& frag_fp) {
	if (m_is_init) return nullptr;
    // m_window_res = {resw, resh};
	m_glfw_window = init_glfw(resw, resh);
	init_opengl_shaders(frag_fp);
    init_imgui();
	Display::m_is_init = true;
	return m_glfw_window;
}

GLFWwindow* Display::init_glfw(int resw, int resh) {
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit()) {
        throw std::runtime_error{"GLFW could not be initialized."};
    }
    
    std::string title = "Synthetic Object NeRF Engine";
    m_glfw_window = glfwCreateWindow(resw, resh, title.c_str(), NULL, NULL);
    if (m_glfw_window == NULL) {
        throw std::runtime_error{"GLFW window could not be created."};
    }
    glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
    if (gl3wInit()) {
        throw std::runtime_error{"GL3W could not be initialized."};
    }
#else
    glewExperimental = 1;
    if (glewInit()) {
        throw std::runtime_error{"GLEW could not be initialized."};
    }
#endif
    glfwSwapInterval(0); // Disable vsync

    GLint gl_version_minor, gl_version_major;
    glGetIntegerv(GL_MINOR_VERSION, &gl_version_minor);
    glGetIntegerv(GL_MAJOR_VERSION, &gl_version_major);

    if (gl_version_major < 3 || (gl_version_major == 3 && gl_version_minor < 1)) {
        throw std::runtime_error{fmt::format("Unsupported OpenGL version {}.{}. instant-ngp requires at least OpenGL 3.1", gl_version_major, gl_version_minor)};
    }

    tlog::success() << "Initialized OpenGL version " << glGetString(GL_VERSION);

	// init_opengl_shaders();

	return m_glfw_window;
}

void Display::init_imgui() {
	float xscale, yscale;
	glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;

	// By default, imgui places its configuration (state of the GUI -- size of windows,
	// which regions are expanded, etc.) in ./imgui.ini relative to the working directory.
	// Instead, we would like to place imgui.ini in the directory that instant-ngp project
	// resides in.
	static std::string ini_filename;
	ini_filename = (File::get_root_dir()/"imgui.ini").str();
	// ini_filename = "./imgui.ini";
	io.IniFilename = ini_filename.c_str();

	// New ImGui event handling seems to make camera controls laggy if input trickling is true.
	// So disable input trickling.
	io.ConfigInputTrickleEventQueue = false;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 140");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);
}

void Display::init_opengl_shaders(const std::string& frag_fp) {
	static const char* shader_vert = R"glsl(#version 140
		out vec2 UVs;
		void main() {
			UVs = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);
			gl_Position = vec4(UVs * 2.0 - 1.0, 0.0, 1.0);
		})glsl";
	
	GLuint vert = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vert, 1, &shader_vert, NULL);
	glCompileShader(vert);
	ngp::check_shader(vert, "Blit vertex shader", false);

	std::string shader_frag_s = File::read_text(frag_fp);

	GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
	const GLchar *shader_frag[] = { shader_frag_s.c_str() };
	glShaderSource(frag, 1, shader_frag, NULL);
	glCompileShader(frag);
	ngp::check_shader(frag, "Blit fragment shader", false);

	m_blit_program = glCreateProgram();
	glAttachShader(m_blit_program, vert);
	glAttachShader(m_blit_program, frag);
	glLinkProgram(m_blit_program);
	ngp::check_shader(m_blit_program, "Blit shader program", true);

	glDeleteShader(vert);
	glDeleteShader(frag);

	glGenVertexArrays(1, &m_blit_vao);
}


void Display::begin_frame() {
	if (glfwWindowShouldClose(m_glfw_window) || ImGui::IsKeyPressed(GLFW_KEY_ESCAPE) || ImGui::IsKeyPressed(GLFW_KEY_Q)) {
		destroy();
		return;
	}

	glfwPollEvents();

	// UI begin
	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
	ImGuizmo::BeginFrame();
}

void Display::transfer_texture(const Foveation& foveation, [[maybe_unused]] GLint syn_rgba, GLint syn_depth, GLint rgba_filter_mode, 
	GLint nerf_rgba, GLint nerf_depth, GLint framebuffer, const ivec2& offset, const ivec2& resolution) {
	if (m_blit_program == 0) {
		return;
	}

	bool tex = glIsEnabled(GL_TEXTURE_2D);
	bool depth = glIsEnabled(GL_DEPTH_TEST);
	bool cull = glIsEnabled(GL_CULL_FACE);

	if (!tex) 
		glEnable(GL_TEXTURE_2D);
	if (!depth) 
		glEnable(GL_DEPTH_TEST);
	if (cull) 
		glDisable(GL_CULL_FACE);

	glDepthFunc(GL_ALWAYS);
	glDepthMask(GL_TRUE);

	glBindVertexArray(m_blit_vao);
	glUseProgram(m_blit_program);
	auto syn_rgba_uniform = glGetUniformLocation(m_blit_program, "syn_rgba");
	auto syn_depth_uniform = glGetUniformLocation(m_blit_program, "syn_depth");
	glUniform1i(syn_rgba_uniform, 0);
	glUniform1i(syn_depth_uniform, 1);
	auto nerf_rgba_uniform = glGetUniformLocation(m_blit_program, "nerf_rgba");
	auto nerf_depth_uniform = glGetUniformLocation(m_blit_program, "nerf_depth");
	glUniform1i(nerf_rgba_uniform, 2);
	glUniform1i(nerf_depth_uniform, 3);

	auto bind_warp = [&](const ngp::FoveationPiecewiseQuadratic& warp, const std::string& uniform_name) {
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".al").c_str()), warp.al);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bl").c_str()), warp.bl);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cl").c_str()), warp.cl);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".am").c_str()), warp.am);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bm").c_str()), warp.bm);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".ar").c_str()), warp.ar);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".br").c_str()), warp.br);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cr").c_str()), warp.cr);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_left").c_str()), warp.switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_right").c_str()), warp.switch_right);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_left").c_str()), warp.inv_switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_right").c_str()), warp.inv_switch_right);
	};

	bind_warp(foveation.warp_x, "warp_x");
	bind_warp(foveation.warp_y, "warp_y");

	glActiveTexture(GL_TEXTURE3);
	glBindTexture(GL_TEXTURE_2D, nerf_depth);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, rgba_filter_mode);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, rgba_filter_mode);

	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, nerf_rgba);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, syn_depth);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, syn_rgba);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, rgba_filter_mode);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, rgba_filter_mode);

	glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
	glViewport(offset.x, offset.y, resolution.x, resolution.y);

	glDrawArrays(GL_TRIANGLES, 0, 3);

	glBindVertexArray(0);
	glUseProgram(0);

	glDepthFunc(GL_LESS);

	// restore old state
	if (!tex) glDisable(GL_TEXTURE_2D);
	if (!depth) glDisable(GL_DEPTH_TEST);
	if (cull) glEnable(GL_CULL_FACE);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

bool Display::present(GLuint nerf_rgba_texid, GLuint nerf_depth_texid, GLuint syn_rgba_texid, GLuint syn_depth_texid, const ivec2& nerf_extent, const Foveation& fov) {
	if (!m_glfw_window) {
		throw std::runtime_error{"Window must be initialized to be presented."};
	}
	// UI DRAWING
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	glfwGetFramebufferSize(m_glfw_window, &m_window_res.x, &m_window_res.y);

	// IMAGE RENDER
	glViewport(0, 0, m_window_res.x, m_window_res.y);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// glEnable(GL_BLEND);
	// glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
	// glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);

    ivec2 extent = {(int)((float)m_window_res.x / nerf_extent.x), (int)((float)m_window_res.y / nerf_extent.y)};
	ivec2 top_left{0, m_window_res.y - extent.y};
	transfer_texture(fov, syn_rgba_texid, syn_depth_texid, GL_LINEAR, nerf_rgba_texid, nerf_depth_texid, m_framebuffer, top_left, extent);
	glFinish();

	// IMGUI
	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	// Visualizations are only meaningful when rendering a single view
	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();
	return true;
}

void Display::destroy() {
	if (!Display::m_is_init) {
		return;
	}

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(m_glfw_window);
	glfwTerminate();

	m_glfw_window = nullptr;
	m_is_init = false;
}

}