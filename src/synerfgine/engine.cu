#include <synerfgine/cuda_helpers.h>
#include <synerfgine/engine.h>

namespace sng {

Engine::Engine() {
    m_devices.emplace_back(find_cuda_device(), true);
}

void Engine::init(int res_width, int res_height) {
    GLFWwindow* glfw_window = m_display.init_window(res_width, res_height, false);
    glfwSetWindowUserPointer(glfw_window, this);
	glfwSetWindowSizeCallback(glfw_window, [](GLFWwindow* window, int width, int height) {
		Engine* engine = (Engine*)glfwGetWindowUserPointer(window);
		if (engine) {
			engine->redraw_next_frame();
		}
	});
}

void Engine::load_file(fs::path path) {
    m_nerf_world.load_network(path);
}

bool Engine::frame() {
    if (m_devices.empty()) {
        tlog::error("No CUDA devices found or attached.");
        return false;
    }

    auto& device = m_devices.front();
    if (!m_display.begin_frame(device, is_dirty)) return false;

    SyncedMultiStream synced_streams{m_stream.get(), 2};
    std::vector<std::future<void>> futures(2);
    auto render_buffer = m_display.get_render_buffer();
    render_buffer->set_color_space(ngp::EColorSpace::SRGB);
    render_buffer->set_tonemap_curve(ngp::ETonemapCurve::Identity);

    futures[0] = device.enqueue_task([this, &device, render_buffer, stream=synced_streams.get(0)]() {
        auto device_guard = use_device(stream, *render_buffer, device);
        m_syn_world.handle(device, m_display.get_window_res());
    });

    futures[1] = device.enqueue_task([this, &device, render_buffer, stream=synced_streams.get(1)]() {
        auto device_guard = use_device(stream, *render_buffer, device);
        m_nerf_world.handle(device, m_display.get_window_res());
    });

    for (auto& future : futures) {
        future.get();
    }

    {
        auto device_guard = use_device(synced_streams.get(0), *render_buffer, device);
        m_display.present(device, m_syn_world);
        m_display.end_frame();
    }

    return true;
}

Engine::~Engine() {
    for (auto&& device : m_devices) {
        device.clear();
    }
}

}