#include "hip/hip_runtime.h"
#include <synerfgine/engine.cuh>
#include <synerfgine/common.cuh>
#include <filesystem/path.h>
#include <iostream>
#include <type_traits>

namespace sng {

void Engine::set_virtual_world(const std::string& config_fp) {
    nlohmann::json config = File::read_json(config_fp);
    nlohmann::json& mat_conf = config["materials"];
    for (uint32_t i = 0; i < mat_conf.size(); ++i) {
        m_materials.emplace_back(i, mat_conf[i]);
    }
    for (const auto& m: m_materials ) {
        m.print();
    }
    nlohmann::json& obj_conf = config["objfile"];
    // init_objs(obj_conf);
}

void Engine::init(int res_width, int res_height, const std::string& frag_fp, Testbed* nerf) {
	m_testbed = nerf;
    m_testbed->m_train = false;
    m_testbed->set_n_views(1);
    m_testbed->m_views.front().visualized_dimension = -1;
    m_testbed->m_views.front().device = &(m_testbed->primary_device());
    m_next_frame_resolution = {res_width, res_height};
    GLFWwindow* glfw_window = m_display.init_window(res_width, res_height, frag_fp);
    glfwSetWindowUserPointer(glfw_window, this);
	glfwSetWindowSizeCallback(glfw_window, [](GLFWwindow* window, int width, int height) {
		Engine* engine = (Engine*)glfwGetWindowUserPointer(window);
		if (engine) {
            engine->m_next_frame_resolution = {width, height};
			engine->redraw_next_frame();
		}
	});
    glfwSetWindowCloseCallback(glfw_window, [](GLFWwindow* window) {
		Engine* engine = (Engine*)glfwGetWindowUserPointer(window);
		if (engine) { engine->set_dead(); }
	});
    Testbed::CudaDevice& device = m_testbed->primary_device();
    m_stream_id = device.stream();
}

void Engine::try_resize() {
    ivec2 curr_window_res = m_display.get_window_res();
    if (curr_window_res != m_next_frame_resolution) {
        m_display.set_window_res(m_next_frame_resolution);
        m_testbed->m_window_res = m_next_frame_resolution;
        auto& view = nerf_render_buffer_view();
        auto nerf_view = view.render_buffer->view();
        nerf_view.hidden_area_mask = nullptr;
        uint32_t nerf_res = product(nerf_view.resolution);
        uint32_t n_pixels_full_res = product(curr_window_res);
		float pixel_ratio = ((float)nerf_res / (float)n_pixels_full_res);
		float last_factor = std::sqrt(pixel_ratio);
		float factor = std::sqrt(pixel_ratio / m_render_ms * 1000.0f / m_testbed->m_dynamic_res_target_fps);
        auto new_res = downscale_resolution(m_next_frame_resolution, factor);
        view.resize(new_res);
        m_testbed->m_views.front().resize(view.full_resolution);
    }
}

bool Engine::frame() {
    if (!m_display.is_alive()) return false;
    Testbed::CudaDevice& device = m_testbed->primary_device();
    device.device_guard();
	m_display.begin_frame();
    try_resize();
    sync();
    m_testbed->handle_user_input();
    m_testbed->apply_camera_smoothing(__timer.get_ave_time("nerf"));

    auto& view = nerf_render_buffer_view();
    view.full_resolution = m_testbed->m_window_res;
    view.camera0 = m_testbed->m_smoothed_camera;
    // Motion blur over the fraction of time that the shutter is open. Interpolate in log-space to preserve rotations.
    view.camera1 = m_testbed->m_camera_path.rendering ? camera_log_lerp(m_testbed->m_smoothed_camera, m_testbed->m_camera_path.render_frame_end_camera, m_testbed->m_camera_path.render_settings.shutter_fraction) : view.camera0;
    view.visualized_dimension = m_testbed->m_visualized_dimension;
    view.relative_focal_length = m_testbed->m_relative_focal_length;
    view.screen_center = m_testbed->m_screen_center;
    view.render_buffer->set_hidden_area_mask(nullptr);
    view.foveation = {};

    auto nerf_view = view.render_buffer->view();
	vec2 focal_length = m_testbed->calc_focal_length(
        nerf_view.resolution, 
        m_testbed->m_relative_focal_length, 
        m_testbed->m_fov_axis, 
        m_testbed->m_zoom);
	vec2 screen_center = m_testbed->render_screen_center(view.screen_center);
    __timer.reset();
    {
        sync();
        m_testbed->primary_device().set_render_buffer_view(nerf_view);
        if (m_testbed->primary_device().dirty()) {
            m_testbed->reset_accumulation(false);
            nerf_view.clear(m_stream_id);
        }
        m_testbed->render_frame(
            m_stream_id,
            view.camera0,
            view.camera1,
            view.prev_camera,
            screen_center,
            view.relative_focal_length,
            view.rolling_shutter,
            view.foveation,
            view.prev_foveation,
            view.visualized_dimension,
            *view.render_buffer
        );
        sync();
        view.prev_camera = view.camera0;
        view.prev_foveation = view.foveation;

        ivec2 nerf_res = nerf_view.resolution;
        auto n_elements = product(nerf_res);
        // linear_kernel(debug_shade, 0, m_stream_id, n_elements, nerf_view.frame_buffer, vec3(1.0, 0.0, 0.0), nerf_view.depth_buffer, 0.0);
        // linear_kernel(print_shade, 0, m_stream_id, n_elements, nerf_view.frame_buffer, nerf_view.depth_buffer);
        // sync();
    }
    m_render_ms = (float)__timer.log_time("nerf");
    m_testbed->m_rgba_render_textures.front()->load_gpu(nerf_view.frame_buffer, nerf_view.resolution, m_nerf_rgba_cpu);
    m_testbed->m_depth_render_textures.front()->load_gpu(nerf_view.depth_buffer, nerf_view.resolution, 1, m_nerf_depth_cpu);
    GLuint nerf_rgba_texid = m_testbed->m_rgba_render_textures.front()->texture();
    GLuint nerf_depth_texid = m_testbed->m_depth_render_textures.front()->texture();

	ImDrawList* list = ImGui::GetBackgroundDrawList();
    m_testbed->draw_visualizations(list, m_testbed->m_smoothed_camera);
    m_display.present(nerf_rgba_texid, nerf_depth_texid, m_testbed->m_n_views(0), view.foveation);

    return m_display.is_alive();
}

}