#include <synerfgine/engine.h>
#include <iostream>

namespace sng {

Engine::Engine() {
    m_devices.emplace_back(find_cuda_device(), true);
}

void Engine::init(int res_width, int res_height, Testbed* nerf) {
    GLFWwindow* glfw_window = m_display.init_window(res_width, res_height, false);
    glfwSetWindowUserPointer(glfw_window, this);
	glfwSetWindowSizeCallback(glfw_window, [](GLFWwindow* window, int width, int height) {
		Engine* engine = (Engine*)glfwGetWindowUserPointer(window);
		if (engine) {
			engine->redraw_next_frame();
		}
	});
    m_nerf_world.init(nerf);
    m_syn_world.mut_camera().set_default_matrix(nerf->m_camera);
	m_testbed = nerf;
    m_testbed->m_train = false;
}

void Engine::debug_rays() {
    auto& device = m_devices.front();
    m_syn_world.handle_user_input(m_display.get_window_res());
    std::shared_ptr<CudaRenderBuffer> render_buffer = m_syn_world.render_buffer();
    render_buffer->set_color_space(ngp::EColorSpace::SRGB);
    render_buffer->set_tonemap_curve(ngp::ETonemapCurve::Identity);
    if (m_show_nerf) {
        m_nerf_world.debug_init_rays(device, m_display.get_window_res(), m_syn_world.camera());
        m_syn_world.reset_frame(device, m_display.get_window_res());
    } else {
        m_syn_world.debug_init_rays(device, m_display.get_window_res());
        m_nerf_world.reset_frame(device, m_display.get_window_res());
    }
}

bool Engine::frame() {
    if (m_devices.empty()) {
        tlog::error("No CUDA devices found or attached.");
        return false;
    }

    auto& device = m_devices.front();
    is_dirty = m_syn_world.handle_user_input(m_display.get_window_res());

    if (!m_display.begin_frame(device, is_dirty)) return false;

    // ImGui::Begin("Debug");
    // if (ImGui::CollapsingHeader("Toggle Buffer Views", ImGuiTreeNodeFlags_DefaultOpen)) {
    //     if (ImGui::RadioButton("Show ray dir buffers", m_is_debug_rays)) {
    //         m_is_debug_rays = !m_is_debug_rays;
    //     }
    //     if (m_is_debug_rays && ImGui::RadioButton("Rays Nerf (Syn when off)", m_show_nerf)) {
    //         m_show_nerf = !m_show_nerf;
    //     }
    // }
    // if (m_is_debug_rays) { 
    //     debug_rays(); 
    //     ImGui::End();
    //     m_display.present(device, m_syn_world, m_nerf_world);
    //     m_display.end_frame();
    //     return true;
    // }

    // ImGui::End();

    {
        SyncedMultiStream synced_streams{m_stream.get(), 3};
        std::vector<std::future<void>> futures(3);

        futures[0] = device.enqueue_task([this, &device, stream=synced_streams.get(0)]() {
            std::shared_ptr<CudaRenderBuffer> render_buffer = m_syn_world.render_buffer();
            render_buffer->set_color_space(ngp::EColorSpace::SRGB);
            render_buffer->set_tonemap_curve(ngp::ETonemapCurve::Identity);
            m_syn_world.handle(device, m_display.get_window_res());
            m_syn_world.shoot_network(device, m_display.get_window_res(), *m_testbed);
            m_syn_world.debug_visualize_pos(device, m_syn_world.sun_pos(), vec3(0.0f, 1.0f, 0.0f), 0.2f);
        });

        futures[1] = device.enqueue_task([this, &device, stream=synced_streams.get(1)]() {
            std::shared_ptr<CudaRenderBuffer> render_buffer = m_nerf_world.render_buffer();
            auto device_guard = use_device(stream, *render_buffer, device);
            m_nerf_world.handle(device, m_syn_world.camera(), 
                m_syn_world.sun(), 
                m_syn_world.get_object(), 
                m_display.get_window_res());
        });

        if (futures[0].valid()) {
            futures[0].get();
        }

        if (futures[1].valid()) {
            futures[1].get(); 
        }

        m_display.present(device, m_syn_world, m_nerf_world);
        m_display.end_frame();
    }

    return true;
}

Engine::~Engine() {
    for (auto&& device : m_devices) {
        device.clear();
    }
}

}